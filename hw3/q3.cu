#include "hip/hip_runtime.h"
// Adam Kraus
// CSC 440
// Homework 3 due 3/25/2024
// Question 3

__global__ void transpose(matrix) {
	// i row, j col

	int i = threadIdx.x / blockDim.x + blockDim.x * blockIdx.x
	int j = threadIdx.y % blockDim.y + blockDim.y * blockIdx.y

	// on diagonal or above
	if(j >= i) return;

	swap(matrix[i][j], matrix[j][i]);
}

