#include "hip/hip_runtime.h"
// Adam Kraus
// CSC 440
// Homework 3 due 3/25/2024
// Question 5

#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

const int k = 20;

__device__ void Merge(unsigned long long* a, unsigned long long* temp, unsigned long long left, unsigned long long middle, unsigned long long right) {
    unsigned long long i = left;
    unsigned long long j = middle;
    unsigned long long k = left;

    while (i < middle && j < right) 
    {
        if (a[i] <= a[j])
            temp[k++] = a[i++];
        else
            temp[k++] = a[j++];
    }

    while (i < middle)
        temp[k++] = a[i++];
    while (j < right)
        temp[k++] = a[j++];

    for (unsigned long long x = left; x < right; x++)
        a[x] = temp[x];
}

__global__ void sortMerge(unsigned long long *a, unsigned long long *temp, unsigned long long n, unsigned long long w) {
	unsigned long long i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long long left = i * w;
    unsigned long long mid = left + w / 2;
    unsigned long long right = left + w;

    if (left < n && mid < n) 
    {
        Merge(a, temp, left, mid, right);
    }
}

int main() {
	srand(time(NULL));

	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// n = 2^k
	unsigned long long n = 1 << k;

	// allocate a[n]
	unsigned long long *d_a;
	hipMallocManaged(&d_a, n*sizeof(unsigned long long));
	unsigned long long *temp;
	hipMallocManaged(&temp, n*sizeof(unsigned long long));

	// initialize a w/ 0 to n-1
	for(unsigned long long i = 0; i < n; i++) {
		d_a[i] = i;
	}

	// shuffle a
	for(unsigned long long i = 0; i < n-1; i++) {
		unsigned long long j = rand() % (n - i) + i;

		unsigned long long temp_val = d_a[i];
		d_a[i] = d_a[j];
		d_a[j] = temp_val;
	}

	// for(unsigned long long i = 0; i < 100; i++) {
	// 	printf("%llu\n", d_a[i]);
	// }

	unsigned long long blockSize = 1024;
	unsigned long long gridSize = (unsigned long long)ceil(float(n)/(blockSize));

	hipEventRecord(start);

	// sort a on device (merge)
	for(unsigned long long w = 1; w < n; w *= 2) {
		sortMerge<<<blockSize, gridSize>>>(d_a, temp, n, w*2);
	}

	// wait for GPU
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time to sort:  %3.1f ms \n", time);

	// check sorted
	bool sorted = true;
	
	for(unsigned long long i = 0; i < n-1; i++) {
		if(d_a[i] > d_a[i+1]) {
			sorted = false;
			printf("%llu\n", i);
			break;
		}
	}

	hipFree(d_a);
	hipFree(temp);

	printf(sorted ? "Sorted\n" : "Not sorted\n");
}