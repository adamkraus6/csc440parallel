// Adam Kraus
// CSC 440
// Homework 3 due 3/25/2024
// Question 4


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

const int k = 20;

__global__ void sortOddEven(unsigned long long *a, unsigned long long n, unsigned long long phase) {
	unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
	// odd threads work on odd phase, even on even
	if(i+1 < n & (i%2 == phase%2)) {
		if(a[i] > a[i+1]) {
			unsigned long long temp = a[i];
			a[i] = a[i+1];
			a[i+1] = temp;
		}
	}
}

int main() {
	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// n = 2^k
	unsigned long long n = 1 << k;

	// allocate a[n]
	unsigned long long *d_a;
	hipMallocManaged(&d_a, n*sizeof(unsigned long long));

	// initialize a w/ 0 to n-1
	for(unsigned long long i = 0; i < n; i++) {
		d_a[i] = i;
	}

	// shuffle a
	for(unsigned long long i = 0; i < n-1; i++) {
		unsigned long long j = rand() % (n - i) + i;

		unsigned long long temp = d_a[i];
		d_a[i] = d_a[j];
		d_a[j] = temp;
	}

	unsigned long long blockSize = 4096;
	unsigned long long gridSize = (unsigned long long)ceil(float(n)/(blockSize));

	hipEventRecord(start);

	// sort a on device (odd-even)
	for(unsigned long long phase = 0; phase < n; phase++) {
		sortOddEven<<<blockSize, gridSize>>>(d_a, n, phase);
	}

	// wait for GPU
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time to sort: %3.1f ms \n", time);

	// check sorted
	bool sorted = true;
	
	for(unsigned long long i = 0; i < n-1; i++) {
		if(d_a[i] > d_a[i+1]) {
			sorted = false;
		}
	}

	hipFree(d_a);

	printf(sorted ? "Sorted\n" : "Not sorted\n");
}