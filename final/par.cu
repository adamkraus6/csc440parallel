
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define BIN_COUNT 16

const int k = 28;
const int numPerThread = 16;

__global__ void sortIntoBins(double* arr, int* bins) {
	int temp_bins[BIN_COUNT] = {0};
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	index *= numPerThread;

	for(int i = index; i < index + numPerThread; i++) {
		int bin = (int)arr[i] % BIN_COUNT;
		temp_bins[bin]++;
	}

	for(int i = 0; i < BIN_COUNT; i++) {
		atomicAdd(&bins[i], temp_bins[i]);
	}

}

double randDouble(double min, double max)
{
    double range = (max - min); 
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

int main() {
	srand(time(NULL));

	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	int n = 1 << k;

	// arrays
	double* arr;
	hipMallocManaged(&arr, n*sizeof(double));
	int* bins;
	hipMallocManaged(&bins, BIN_COUNT*sizeof(int));

	// init array with random values
	for(int i = 0; i < n; i++) {
		// max of 4194304
		arr[i] = randDouble(0, 1<<22);
	}

	// zero bins array
	for(int i = 0; i < BIN_COUNT; i++) {
		bins[i] = 0;
	}

	// play with these numbers
	int blockSize = 1024;
	int gridSize = (int)ceil(float(n/numPerThread)/blockSize);

	// printf("blockSize: %d\ngridSize: %d\n", blockSize, gridSize);

	hipEventRecord(start);

	// sort array into bins
	sortIntoBins<<<gridSize, blockSize>>>(arr, bins);

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time to sort: %.7f seconds\n", time/1000);

	int sum = 0;

	// print out number in each bin
	for(int i = 0; i < BIN_COUNT; i++) {
		printf("Bin %d: %d\n", i+1, bins[i]);
		sum += bins[i];
	}

	printf("Total bin count: %d (Should be %d)\n", sum, n);

	hipFree(arr);
	hipFree(bins);
}