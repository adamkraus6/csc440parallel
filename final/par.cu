
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

const int k = 10;

__global__ void sortIntoBins() {
	return;
}

double randDouble(double min, double max)
{
    double range = (max - min); 
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

int main() {
	srand(time(NULL));

	int n = 1 << k;
	int bin_count = 1 << 4; // 16 bins

	// arrays
	double* arr;
	hipMallocManaged(&arr, n*sizeof(double));
	int* bins;
	hipMallocManaged(&bins, bin_count*sizeof(int));

	// init array with random values
	for(int i = 0; i < n; i++) {
		// max of 4194304
		arr[i] = randDouble(0, 1<<22);
	}

	// zero bins array
	for(int i = 0; i < bin_count; i++) {
		bins[i] = 0;
	}

	// test print array
	// for(int i = 0; i < n; i++) {
	// 	printf("%.4f\n", arr[i]);
	// }

	int blockSize = 1024;
	int gridSize = (int)ceil(float(n)/blockSize);

	// sort array into bins
	// sortIntoBins<<<blockSize, gridSize>>>();

	int sum = 0;

	// print out number in each bin
	for(int i = 0; i < bin_count; i++) {
		printf("Bin %d: %d\n", i+1, bins[i]);
		sum += bins[i];
	}

	printf("Total bin count: %d (Should be %d)\n", sum, n);
}